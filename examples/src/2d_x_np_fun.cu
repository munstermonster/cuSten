#include "hip/hip_runtime.h"
// Andrew Gloster
// May 2018
// Example of x direction non periodic 2D code

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.


// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <cmath>
#include <iostream>
#include <cstdio>
#include "hip/hip_runtime.h"

// ---------------------------------------------------------------------
// cuSten - Note the file position is relative
// ---------------------------------------------------------------------

#include "../../cuSten/cuSten.h"

// ---------------------------------------------------------------------
// MACROS
// ---------------------------------------------------------------------

#define BLOCK_X 32
#define BLOCK_Y 32

// ---------------------------------------------------------------------
// Function pointer definition
// ---------------------------------------------------------------------

// Data -- Coefficients -- Stencil Centre Index
typedef double (*devArg1X)(double*, double*, int);

__inline__ __device__ double CentralDifference(double* data, double* coe, int loc)
{	
	return (data[loc - 1] - 2 * data[loc] + data[loc + 1]) * coe[0];	
}

__device__ devArg1X devfunc = CentralDifference;

// ---------------------------------------------------------------------
// Main Program
// ---------------------------------------------------------------------

int main()
{	
	// Set the device number
	int deviceNum = 0;

	// Declare Domain Size
	int nx = 8192;
	int ny = 8192;

	double lx = 2 * M_PI;

	// Domain spacings
	double dx = lx / (double) (nx);

	// Set the number of tiles per device
	int numTiles = 4;

	// Initial Conditions
	double* dataOld;
	double* dataNew;
	double* answer;

	// -----------------------------
	// Allocate the memory 
	// -----------------------------

	hipMallocManaged(&dataOld, nx * ny * sizeof(double));
	hipMallocManaged(&dataNew, nx * ny * sizeof(double));
	hipMallocManaged(&answer, nx * ny * sizeof(double));

	// -----------------------------
	// Set the initial conditions
	// -----------------------------

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			dataOld[j * nx + i] = sin(i * dx);
			dataNew[j * nx + i] = 0.0;
			answer[j * nx + i] =- sin(i * dx);
		}
	}


	// // Ensure all the above is completed
	hipDeviceSynchronize();

	// -----------------------------
	// Set the stencil to compute
	// -----------------------------

	int numSten = 3;
	int numStenLeft = 1;
	int numStenRight = 1;

	int numCoe = 1;

	double* coe;
	hipMallocManaged(&coe, numCoe * sizeof(double));

	coe[0] = 1.0 / pow(dx, 2.0);

	// -----------------------------
	// Set up device
	// -----------------------------

	// Set up the compute device structs
	cuSten_t<double> xDirCompute;

	// Copy the function pointer to the device
	double* func;
	hipMemcpyFromSymbol(&func, HIP_SYMBOL(devfunc), sizeof(devArg1X));

	// Initialise the instance of the stencil
	cuStenCreate2DXnpFun(&xDirCompute, deviceNum, numTiles, nx, ny, BLOCK_X, BLOCK_Y, dataNew, dataOld, coe, numSten, numStenLeft, numStenRight, numCoe, func);

	// Synchronise to ensure everything initialised
	hipDeviceSynchronize();

	// -----------------------------
	// Compute
	// -----------------------------
	
	// Run the computation
	cuStenCompute2DXnpFun(&xDirCompute, HOST);

	// Synchronise at the end to ensure everything is complete
	hipDeviceSynchronize();

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			printf("%lf %lf %d \n", dataNew[j * nx + i], answer[j * nx + i], i);
		}
	}

	// -----------------------------
	// Destroy struct and free memory
	// -----------------------------

	// Destroy struct
	cuStenDestroy2DXnpFun(&xDirCompute);

	// Free memory at the end
	hipFree(dataOld);
	hipFree(dataNew);
	hipFree(answer);
	hipFree(coe);
	
	// Return 0 when the program completes
	return 0;
}
