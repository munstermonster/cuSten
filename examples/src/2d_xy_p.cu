#include "hip/hip_runtime.h"
// Andrew Gloster
// May 2018
// Example of xy direction periodic 2D code

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.


// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <cmath>
#include <iostream>
#include <cstdio>
#include "hip/hip_runtime.h"

// ---------------------------------------------------------------------
// cuSten - Note the file position is relative
// ---------------------------------------------------------------------

#include "../../cuSten/cuSten.h"

// ---------------------------------------------------------------------
// MACROS
// ---------------------------------------------------------------------

#define BLOCK_X 32
#define BLOCK_Y 32

// ---------------------------------------------------------------------
// Main Program
// ---------------------------------------------------------------------

int main()
{	
	// Set the device number
	int deviceNum = 0;

	// Declare Domain Size
	int nx = 8192;
	int ny = 8192;

	double lx = 2 * M_PI;
	double ly = 2 * M_PI;

	// Domain spacings
	double dx = lx / (double) (nx);
	double dy = ly / (double) (ny);

	// Set the number of tiles per device
	int numTiles = 1;

	// Initial Conditions
	double* dataInput;
	double* dataOutput;
	double* answer;

	// -----------------------------
	// Allocate the memory 
	// -----------------------------

	hipMallocManaged(&dataInput, nx * ny * sizeof(double));
	hipMallocManaged(&dataOutput, nx * ny * sizeof(double));
	hipMallocManaged(&answer, nx * ny * sizeof(double));

	// -----------------------------
	// Set the initial conditions
	// -----------------------------

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			dataInput[j * nx + i] = sin(i * dx) * cos(j * dy);
			dataOutput[j * nx + i] = 0.0;
			answer[j * nx + i] = - cos(i * dx) * sin(j * dy);
		}
	}

	// Ensure all the above is completed
	hipDeviceSynchronize();

	// -----------------------------
	// Set the stencil to compute
	// -----------------------------

	int numStenHoriz = 3;
	int numStenLeft = 1;
	int numStenRight = 1;

	int numStenVert = 3;
	int numStenTop = 1;
	int numStenBottom = 1;

	double* weights;
	hipMallocManaged(&weights, numStenHoriz * numStenVert * sizeof(double));

	double sigma = 1.0 / (4.0 * dx * dy);

	weights[0] = 1.0 * sigma;
	weights[1] = 0.0 * sigma;
	weights[2] = - 1.0 * sigma;
	weights[3] = 0.0 * sigma;
	weights[4] = 0.0 * sigma;
	weights[5] = 0.0 * sigma;
	weights[6] = - 1.0 * sigma;
	weights[7] = 0.0 * sigma;
	weights[8] = 1.0 * sigma;

	// -----------------------------
	// Set up device
	// -----------------------------

	// Set up the compute device structs
	cuSten_t<double> xyDirCompute;

	// Initialise the instance of the stencil
	cuStenCreate2DXYp(
		&xyDirCompute,

		deviceNum,

		numTiles,

		nx,
		ny,

		BLOCK_X,
		BLOCK_Y,

		dataOutput,
		dataInput,
		weights,

		numStenHoriz,
		numStenLeft,
		numStenRight,

		numStenVert,
		numStenTop,
		numStenBottom
	);

	// Synchronise to ensure everything initialised
	hipDeviceSynchronize();

	// -----------------------------
	// Compute
	// -----------------------------

	// Run the computation
	cuStenCompute2DXYp(&xyDirCompute, HOST);

	// // Synchronise at the end to ensure everything is complete
	hipDeviceSynchronize();

	// -----------------------------
	// Destroy struct and free memory
	// -----------------------------

	// Destroy struct
	cuStenDestroy2DXYp(&xyDirCompute);

	// Free memory at the end
	hipFree(dataInput);
	hipFree(dataOutput);
	hipFree(answer);
	hipFree(weights);
	
	// Return 0 when the program completes
	return 0;
}
