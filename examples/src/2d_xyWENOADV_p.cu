#include "hip/hip_runtime.h"
// Andrew Gloster
// November 2018
// Example of advection in 2D with upwinding WENO

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.


// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <cmath>
#include <iostream>
#include <cstdio>
#include "hip/hip_runtime.h"

// ---------------------------------------------------------------------
// cuSten - Note the file position is relative
// ---------------------------------------------------------------------

#include "../../cuSten/cuSten.h"

// ---------------------------------------------------------------------
// MACROS
// ---------------------------------------------------------------------

#define BLOCK_X 32
#define BLOCK_Y 32

// ---------------------------------------------------------------------
// Main Program
// ---------------------------------------------------------------------

int main()
{	
	// Set the device number
	int deviceNum = 0;

	// Declare Domain Size
	int nx = 8192;
	int ny = 8192;

	double lx = 2 * M_PI;
	double ly = 2 * M_PI;

	// Domain spacings
	double dx = lx / (double) (nx);
	double dy = ly / (double) (ny);

	// Set the number of tiles per device
	int numTiles = 4;

	// Initial Conditions
	double* dataInput;
	double* dataOutput;
	double* u;
	double* v;

	// -----------------------------
	// Allocate the memory 
	// -----------------------------

	hipMallocManaged(&dataInput, nx * ny * sizeof(double));
	hipMallocManaged(&dataOutput, nx * ny * sizeof(double));

	hipMallocManaged(&u, nx * ny * sizeof(double));
	hipMallocManaged(&v, nx * ny * sizeof(double));

	// -----------------------------
	// Set the initial conditions
	// -----------------------------

	// Indexing
	int temp;
	int index;

	for (int j = 0; j < ny; j++)
	{
		temp = j * nx;

		for (int i = 0; i < nx; i++)
		{
			index = temp + i;

			dataInput[index] = cos(i * dx) * sin(j * dy);
			dataOutput[index] = 0.0;

			u[index] = sin(j * dy);
			v[index] = - sin(i * dx);
		}
	}

	// Ensure all the above is completed
	hipDeviceSynchronize();

	// -----------------------------
	// Set up device
	// -----------------------------

	// Set up the compute device structs
	cuSten_t<double> xyWENOCompute;

	// Initialise the instance of the stencil
	cuStenCreate2DXYWENOADVp(	
		&xyWENOCompute,

		deviceNum,

		numTiles,

		nx,
		ny,

		BLOCK_X,
		BLOCK_Y,

		dx,
		dy,

		u,
		v,

		dataOutput,

		dataInput
	);

	// Synchronise to ensure everything initialised
	hipDeviceSynchronize();

	// -----------------------------
	// Compute
	// -----------------------------

	// Run the computation
	cuStenCompute2DXYWENOADVp(&xyWENOCompute, HOST);

	// // Synchronise at the end to ensure everything is complete
	hipDeviceSynchronize();

	// -----------------------------
	// Destroy struct and free memory
	// -----------------------------

	// Destroy struct
	cuStenDestroy2DXYWENOADVp(&xyWENOCompute);

	// Free memory at the end
	hipFree(dataInput);
	hipFree(dataOutput);

	hipFree(u);
	hipFree(v);
	
	// Return 0 when the program completes
	return 0;
}
