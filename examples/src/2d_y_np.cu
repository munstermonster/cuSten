#include "hip/hip_runtime.h"
// Andrew Gloster
// July 2018
// Examples - 2D y direction - non periodic

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.


// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <cmath>
#include <iostream>
#include <cstdio>
#include "hip/hip_runtime.h"

// ---------------------------------------------------------------------
// cuSten - Note the file position is relative
// ---------------------------------------------------------------------

#include "../../cuSten/cuSten.h"

// ---------------------------------------------------------------------
// MACROS
// ---------------------------------------------------------------------

#define BLOCK_X 8
#define BLOCK_Y 8

// ---------------------------------------------------------------------
// Main Program
// ---------------------------------------------------------------------

int main()
{	
	// Set the device number
	int deviceNum = 0;

	// Declare Domain Size
	int nx = 512;
	int ny = 512;

	double ly = 2 * M_PI;

	// Domain spacings
	double dy = ly / (double) (ny);

	// Set the number of tiles per device
	int numTiles = 2;

	// Initial Conditions
	double* dataOld;
	double* dataNew;
	double* answer;

	// -----------------------------
	// Allocate the memory 
	// -----------------------------

	hipMallocManaged(&dataOld, nx * ny * sizeof(double));
	hipMallocManaged(&dataNew, nx * ny * sizeof(double));
	hipMallocManaged(&answer, nx * ny * sizeof(double));

	// -----------------------------
	// Set the initial conditions
	// -----------------------------

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			dataOld[j * nx + i] = sin(j * dy);
			dataNew[j * nx + i] = 0.0;
			answer[j * nx + i] =- sin(j * dy);
		}
	}


	// // Ensure all the above is completed
	hipDeviceSynchronize();

	// -----------------------------
	// Set the stencil to compute
	// -----------------------------

	int numSten = 9;
	int numStenTop = 4;
	int numStenBottom = 4;

	double* weights;
	hipMallocManaged(&weights, numSten * sizeof(double));

	weights[0] = - (1.0 / 560.0) * 1.0 / pow(dy, 2.0);
	weights[1] = (8.0 / 315.0) * 1.0 / pow(dy, 2.0);
	weights[2] = - (1.0 / 5.0) * 1.0 / pow(dy, 2.0);
	weights[3] = (8.0 / 5.0) * 1.0 / pow(dy, 2.0);
	weights[4] = - (205.0 / 72.0) * 1.0 / pow(dy, 2.0);
	weights[5] = (8.0 / 5.0) * 1.0 / pow(dy, 2.0);
	weights[6] = - (1.0 / 5.0) * 1.0 / pow(dy, 2.0);
	weights[7] = (8.0 / 315.0) * 1.0 / pow(dy, 2.0);
	weights[8] = - (1.0 / 560.0) * 1.0 / pow(dy, 2.0);
	// -----------------------------
	// Set up device
	// -----------------------------

	// Set up the compute device structs
	cuSten_t<double> yDirCompute;

	// Initialise the instance of the stencil
	cuStenCreate2DYnp(&yDirCompute, deviceNum, numTiles, nx, ny, BLOCK_X, BLOCK_Y, dataNew, dataOld, weights, numSten, numStenTop, numStenBottom);

	// Synchronise to ensure everything initialised
	hipDeviceSynchronize();

	// -----------------------------
	// Compute
	// -----------------------------

	// Run the computation
	cuStenCompute2DYnp(&yDirCompute, HOST);

	// Synchronise at the end to ensure everything is complete
	hipDeviceSynchronize();

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			printf("%lf %lf %lf %d %d \n", dataOld[j * nx + i], dataNew[j * nx + i], answer[j * nx + i], i, j);
		}
	}

	// -----------------------------
	// Destroy struct and free memory
	// -----------------------------

	// Destroy struct
	cuStenDestroy2DYpFun(&yDirCompute);

	// Free memory at the end
	hipFree(dataOld);
	hipFree(dataNew);
	hipFree(answer);
	hipFree(weights);
	
	// Return 0 when the program completes
	return 0;
}
