// Andrew Gloster
// November 2018

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.

/*! \file custenCreateDestroy2DXYADVWENOp.cu
    Functions to create and destroy the required struct for a 2D xy WENO
 	periodic calculation
*/

// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <iostream>

// ---------------------------------------------------------------------
// Custom libraries and headers
// ---------------------------------------------------------------------

#include "cuSten_struct_type.h"
#include "cuSten_struct_functions.h"
#include "../util/util.h"

// ---------------------------------------------------------------------
// Function to create the struct
// ---------------------------------------------------------------------

/*! \fun void cuStenCreate2DXYWENOADVp
    \brief Function to set up cuSten_t
    \param pt_cuSten Pointer to cuSten type provided by user
	\param numTiles Number of tiles to divide the data into
	\param nx Total number of points in the x direction 
	\param ny Total number of points in the y direction 
	\param BLOCK_X Size of thread block in the x direction
	\param BLOCK_Y Size of thread block in the y direction
	\param dx Spacing of grid in x direction
	\param dy Spacing of grid in y direction
	\param u Pointer to u velocity data
	\param v Pointer to v velocity data
    \param dataOutput Pointer to data output by the function
	\param dataInput Pointer to data input to the function    
*/

template <typename elemType>
void cuStenCreate2DXYWENOADVp(
	cuSten_t<elemType>* pt_cuSten,
	int deviceNum,
	int numTiles,
	int nx,
	int ny,
	int BLOCK_X,
	int BLOCK_Y,
	elemType dx,
	elemType dy,
	elemType* u,
	elemType* v,
	elemType* dataOutput,
	elemType* dataInput
) 
{
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set the device number associated with the struct
  	pt_cuSten->deviceNum = deviceNum;

  	// Set the number of streams
  	pt_cuSten->numStreams = 6;

  	// Set the number of tiles
  	pt_cuSten->numTiles = numTiles;

  	// Set the number points in x on the device
  	pt_cuSten->nx = nx;

  	// Set the number points in y on the device
  	pt_cuSten->ny = ny;

  	// Number of threads in x on the device
	pt_cuSten->BLOCK_X = BLOCK_X;

  	// Number of threads in y on the device
	pt_cuSten->BLOCK_Y = BLOCK_Y;

	// Encode spacings
	pt_cuSten->coeDx = 1.0 / dx;
	pt_cuSten->coeDy = 1.0 / dy;

	// We require i - 3 throught to i + 3 for WENO
	// Set number of points to the left in the stencil
	pt_cuSten->numStenLeft = 3;

	// Set number of points to the right in the stencil
	pt_cuSten->numStenRight = 3;

	// Set number of points in the top the stencil
	pt_cuSten->numStenTop = 3;

	// Set number of points in the bottom of the stencil
	pt_cuSten->numStenBottom = 3;

	// Set size of stencil
	int numStenHoriz = 7;
	int numStenVert = 7;
	
	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);	

	// Create memeory for the streams
	pt_cuSten->streams = (hipStream_t*)malloc(pt_cuSten->numStreams * sizeof(hipStream_t*));

	// Create the streams
	for (int st = 0; st < pt_cuSten->numStreams; st++)
	{
		hipStreamCreate(&pt_cuSten->streams[st]);
		sprintf(msgStringBuffer, "Creating stream %d on GPU %d", st, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);	
	}

	// Create memeory for the events
	pt_cuSten->events = (hipEvent_t*)malloc(2 * sizeof(hipEvent_t*));

	// Create the events
	for (int ev = 0; ev < 2; ev++)
	{
		hipEventCreate(&pt_cuSten->events[ev]);
		sprintf(msgStringBuffer, "Creating event %d on GPU %d", ev, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);
	}

	// Set number of points in the stencil
	pt_cuSten->numSten = numStenHoriz * numStenVert;

	// Set local block array sizes - x direction
	pt_cuSten->nxLocal = pt_cuSten->BLOCK_X + pt_cuSten->numStenLeft + pt_cuSten->numStenRight;

	// Set loacl block array sizes - y direction
	pt_cuSten->nyLocal = pt_cuSten->BLOCK_Y + pt_cuSten->numStenTop + pt_cuSten->numStenBottom;

	// Set the amount of shared memory required
	pt_cuSten->mem_shared = (pt_cuSten->nxLocal * pt_cuSten->nyLocal) * sizeof(elemType) + pt_cuSten->numSten * sizeof(elemType);

	// Find number of points per tile
	pt_cuSten->nyTile = pt_cuSten->ny / pt_cuSten->numTiles;	

	// Set the grid up
    pt_cuSten->xGrid = (pt_cuSten->nx % pt_cuSten->BLOCK_X == 0) ? (pt_cuSten->nx / pt_cuSten->BLOCK_X) : (pt_cuSten->nx / pt_cuSten->BLOCK_X + 1);
    pt_cuSten->yGrid = (pt_cuSten->nyTile % pt_cuSten->BLOCK_Y == 0) ? (pt_cuSten->nyTile / pt_cuSten->BLOCK_Y) : (pt_cuSten->nyTile / pt_cuSten->BLOCK_Y + 1);

	// Allocate the pointers for each input tile
	pt_cuSten->dataInput = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// Allocate the pointers for each output tile
	pt_cuSten->dataOutput = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// Allocate the pointers for each input x velocity tile
	pt_cuSten->uVel = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// Allocate the pointers for each input v velocity tile
	pt_cuSten->vVel = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// // Tile offset index
	int offset = pt_cuSten->nx * pt_cuSten->nyTile;

	// // Match the pointers to the data
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{	
		// Set the input data
		pt_cuSten->dataInput[tile] = &dataInput[tile * offset];

		// Set the output data
		pt_cuSten->dataOutput[tile] = &dataOutput[tile * offset];

		// Set the u velocity tiles
		pt_cuSten->uVel[tile] = &u[tile * offset];

		// Set the v velocity tiles
		pt_cuSten->vVel[tile] = &v[tile * offset];
	}

	// Create cases depending on what tile numbers - Periodic
	// 1 tile
	// 2 tiles
	// 3 or greater

	// Allocate top boundary memory
	pt_cuSten->boundaryTop = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// Allocate bottom boundary memory
	pt_cuSten->boundaryBottom = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	switch(pt_cuSten->numTiles)
	{
		// One tile only requires single top and bottom to be set
		case 1:
			pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[0] = &dataInput[0]; 

			break;

		// Two tiles requires a special case of only setting two tiles
		case 2:
			pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[0] = &dataInput[pt_cuSten->nyTile * pt_cuSten->nx];

			pt_cuSten->boundaryTop[1] = &dataInput[(pt_cuSten->nyTile - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[1] = &dataInput[0];

			break;

		// Default case has interiors, so set the top tile, then loop over interior, then set the bottom tile
		default:
			pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[0] = &dataInput[pt_cuSten->nyTile * pt_cuSten->nx];

			for (int tile = 1; tile < pt_cuSten->numTiles - 1; tile++)
			{
				pt_cuSten->boundaryTop[tile] = &dataInput[(pt_cuSten->nyTile * tile - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[tile] = &dataInput[(pt_cuSten->nyTile * (tile + 1)) * pt_cuSten->nx];
			}

			pt_cuSten->boundaryTop[pt_cuSten->numTiles - 1] = &dataInput[(pt_cuSten->nyTile * (pt_cuSten->numTiles - 1) - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[pt_cuSten->numTiles - 1] = &dataInput[0];

			break;
	}

	// Number of points in top boundary data
	pt_cuSten->numBoundaryTop = pt_cuSten->numStenTop * pt_cuSten->nx;

	// Number of points in bottom boundary data
	pt_cuSten->numBoundaryBottom = pt_cuSten->numStenBottom * pt_cuSten->nx;

	// Number of points in a horizontal stencil
	pt_cuSten->numStenHoriz = numStenHoriz;

	// Number of points in a vertical stencil
	pt_cuSten->numStenVert = numStenVert;
}

// ---------------------------------------------------------------------
// Swap pointers
// ---------------------------------------------------------------------

/*! \fun void cuStenSwap2DXYWENOADVp
    \brief Function to swap pointers necessary for timestepping
    \param pt_cuSten Pointer to cuSten type provided by user
	\param dataInput Pointer to data input to the on the next compute
*/

template <typename elemType>
void cuStenSwap2DXYWENOADVp(
	cuSten_t<elemType>* pt_cuSten,
	elemType* dataInput
) 
{
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{	
		// Swap the input and output data
		std::swap(pt_cuSten->dataInput[tile], pt_cuSten->dataOutput[tile]);

		// Update the boundary data
		switch(pt_cuSten->numTiles)
		{
			// One tile only requires single top and bottom to be set
			case 1:
				pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[0] = &dataInput[0]; 

				break;

			// Two tiles requires a special case of only setting two tiles
			case 2:
				pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[0] = &dataInput[pt_cuSten->nyTile * pt_cuSten->nx];

				pt_cuSten->boundaryTop[1] = &dataInput[(pt_cuSten->nyTile - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[1] = &dataInput[0];

				break;

			// Default case has interiors, so set the top tile, then loop over interior, then set the bottom tile
			default:
				pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[0] = &dataInput[pt_cuSten->nyTile * pt_cuSten->nx];

				for (int tile = 1; tile < pt_cuSten->numTiles - 1; tile++)
				{
					pt_cuSten->boundaryTop[tile] = &dataInput[(pt_cuSten->nyTile * tile - pt_cuSten->numStenTop) * pt_cuSten->nx];
					pt_cuSten->boundaryBottom[tile] = &dataInput[(pt_cuSten->nyTile * (tile + 1)) * pt_cuSten->nx];
				}

				pt_cuSten->boundaryTop[pt_cuSten->numTiles - 1] = &dataInput[(pt_cuSten->nyTile * (pt_cuSten->numTiles - 1) - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[pt_cuSten->numTiles - 1] = &dataInput[0];

				break;
		}
	}
}

// ---------------------------------------------------------------------
// Function to destroy the struct
// ---------------------------------------------------------------------

/*! \fun void cuStenDestroy2DXYWENOADVp
    \brief Function to destroy data associated with cuSten_t
    \param pt_cuSten Pointer to cuSten type provided by user
*/

template <typename elemType>
void cuStenDestroy2DXYWENOADVp
(
	cuSten_t<elemType>* pt_cuSten
) 
{
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);	


	// Destroy the streams
	for (int st = 0; st < pt_cuSten->numStreams; st++)
	{
		hipStreamDestroy(pt_cuSten->streams[st]);
		sprintf(msgStringBuffer, "Destroying stream %d on GPU %d", st, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);	
	}

	// Free the main memory
	free(pt_cuSten->streams);

	// // Create the events
	for (int ev = 0; ev < 2; ev++)
	{
		hipEventDestroy(pt_cuSten->events[ev]);
		sprintf(msgStringBuffer, "Destroying event %d on GPU %d", ev, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);
	}

	// Free the main memory
	free(pt_cuSten->events);

	// Free the pointers for each input tile
	free(pt_cuSten->dataInput);

	// Free the pointers for each output tile
	free(pt_cuSten->dataOutput);

	// Free the top boundary tile pointers
	free(pt_cuSten->boundaryTop);

	// Free the bottom boundary tile pointers
	free(pt_cuSten->boundaryBottom);
}

// ---------------------------------------------------------------------
// Explicit instantiation
// ---------------------------------------------------------------------

template
void cuStenCreate2DXYWENOADVp<double>(
	cuSten_t<double>*,
	int,
	int,
	int,
	int,
	int,
	int,
	double,
	double,
	double*,
	double*,
	double*,
	double*
);

template
void cuStenSwap2DXYWENOADVp<double>(
	cuSten_t<double>*,
	double*
);

template
void cuStenDestroy2DXYWENOADVp<double>(
	cuSten_t<double>*
);

template
void cuStenCreate2DXYWENOADVp<float>(
	cuSten_t<float>*,
	int,
	int,
	int,
	int,
	int,
	int,
	float,
	float,
	float*,
	float*,
	float*,
	float*
);

template
void cuStenSwap2DXYWENOADVp<float>(
	cuSten_t<float>*,
	float*
);

template
void cuStenDestroy2DXYWENOADVp<float>(
	cuSten_t<float>*
);

// ---------------------------------------------------------------------
// End of file
// ---------------------------------------------------------------------
