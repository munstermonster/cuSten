// Andrew Gloster
// May 2018

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.

/*! \file custenCreateDestroy2DYpFun.cu
    Functions to create and destroy the cuSten_t that is used to give input to the compute kernels. 
    2D y direction, periodic
*/

// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <iostream>

// ---------------------------------------------------------------------
// Custom libraries and headers
// ---------------------------------------------------------------------

#include "cuSten_struct_type.h"
#include "cuSten_struct_functions.h"
#include "../util/util.h"

// ---------------------------------------------------------------------
// Function to create the struct
// ---------------------------------------------------------------------

/*! \fun void cuStenCreate2DYp
    \brief Function to set up cuSten_t
    \param pt_cuSten Pointer to cuSten type provided by user
	\param numTiles Number of tiles to divide the data into
	\param nx Total number of points in the x direction 
	\param ny Total number of points in the y direction 
	\param BLOCK_X Size of thread block in the x direction
	\param BLOCK_Y Size of thread block in the y direction
    \param dataOutput Pointer to data output by the function
	\param dataInput Pointer to data input to the function    
	\param weights Pointer to the weights for the stencil
	\param numStenSten Total number of points in the stencil in the y direction
	\param numStenTop Number of points on the top of the stencil
	\param numStenBottom Number of points on the bottom of the stencil
*/

template <typename elemType>
void cuStenCreate2DYp(
	cuSten_t<elemType>* pt_cuSten,
	int deviceNum,
	int numTiles,
	int nx,
	int ny,
	int BLOCK_X,
	int BLOCK_Y,
	elemType* dataOutput,
	elemType* dataInput,
	elemType* weights,
	int numSten,
	int numStenTop,
	int numStenBottom
) 
{
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set the device number associated with the struct
  	pt_cuSten->deviceNum = deviceNum;

  	// Set the number of streams
  	pt_cuSten->numStreams = 3;

  	// Set the number of tiles
  	pt_cuSten->numTiles = numTiles;

  	// Set the number points in x on the device
  	pt_cuSten->nx = nx;

  	// Set the number points in y on the device
  	pt_cuSten->ny = ny;

  	// Number of threads in x on the device
	pt_cuSten->BLOCK_X = BLOCK_X;

  	// Number of threads in y on the device
	pt_cuSten->BLOCK_Y = BLOCK_Y;

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);	

	// Create memeory for the streams
	pt_cuSten->streams = (hipStream_t*)malloc(pt_cuSten->numStreams * sizeof(hipStream_t*));

	// Create the streams
	for (int st = 0; st < pt_cuSten->numStreams; st++)
	{
		hipStreamCreate(&pt_cuSten->streams[st]);
		sprintf(msgStringBuffer, "Creating stream %d on GPU %d", st, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);	
	}

	// Create memeory for the events
	pt_cuSten->events = (hipEvent_t*)malloc(2 * sizeof(hipEvent_t*));

	// Create the events
	for (int ev = 0; ev < 2; ev++)
	{
		hipEventCreate(&pt_cuSten->events[ev]);
		sprintf(msgStringBuffer, "Creating event %d on GPU %d", ev, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);
	}

	// Set number of points in the stencil
	pt_cuSten->numSten = numSten;

	// Set number of points to the left in the stencil
	pt_cuSten->numStenTop = numStenTop;

	// Set number of points to the right in the stencil
	pt_cuSten->numStenBottom = numStenBottom;

	// Set local block array sizes - x direction
	pt_cuSten->nxLocal = pt_cuSten->BLOCK_X;

	// Set loacl block array sizes - y direction
	pt_cuSten->nyLocal = pt_cuSten->BLOCK_Y + pt_cuSten->numStenTop + pt_cuSten->numStenBottom;

	// Set the amount of shared memory required
	pt_cuSten->mem_shared = (pt_cuSten->nxLocal * pt_cuSten->nyLocal) * sizeof(elemType) + pt_cuSten->numSten * sizeof(elemType);

	// Find number of points per tile
	pt_cuSten->nyTile = pt_cuSten->ny / pt_cuSten->numTiles;	

	// Set the grid up
    pt_cuSten->xGrid = (pt_cuSten->nx % pt_cuSten->BLOCK_X == 0) ? (pt_cuSten->nx / pt_cuSten->BLOCK_X) : (pt_cuSten->nx / pt_cuSten->BLOCK_X + 1);
    pt_cuSten->yGrid = (pt_cuSten->nyTile % pt_cuSten->BLOCK_Y == 0) ? (pt_cuSten->nyTile / pt_cuSten->BLOCK_Y) : (pt_cuSten->nyTile / pt_cuSten->BLOCK_Y + 1);

	// Set the device weights pointer
	pt_cuSten->weights = weights;

	// Allocate the pointers for each input tile
	pt_cuSten->dataInput = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// Allocate the pointers for each output tile
	pt_cuSten->dataOutput = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// // Tile offset index
	int offset = pt_cuSten->nx * pt_cuSten->nyTile;

	// // Match the pointers to the data
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{	
		// Set the input data
		pt_cuSten->dataInput[tile] = &dataInput[tile * offset];

		// Set the output data
		pt_cuSten->dataOutput[tile] = &dataOutput[tile * offset];
	}

	// Create cases depending on what tile numbers - Periodic
	// 1 tile
	// 2 tiles
	// 3 or greater

	// Allocate top boundary memory
	pt_cuSten->boundaryTop = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// Allocate bottom boundary memory
	pt_cuSten->boundaryBottom = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	switch(pt_cuSten->numTiles)
	{
		// One tile only requires single top and bottom to be set
		case 1:
			pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[0] = &dataInput[0]; 

			break;

		// Two tiles requires a special case of only setting two tiles
		case 2:
			pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[0] = &dataInput[pt_cuSten->nyTile * pt_cuSten->nx];

			pt_cuSten->boundaryTop[1] = &dataInput[(pt_cuSten->nyTile - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[1] = &dataInput[0];

			break;

		// Default case has interiors, so set the top tile, then loop over interior, then set the bottom tile
		default:
			pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[0] = &dataInput[pt_cuSten->nyTile * pt_cuSten->nx];

			for (int tile = 1; tile < pt_cuSten->numTiles - 1; tile++)
			{
				pt_cuSten->boundaryTop[tile] = &dataInput[(pt_cuSten->nyTile * tile - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[tile] = &dataInput[(pt_cuSten->nyTile * (tile + 1)) * pt_cuSten->nx];
			}

			pt_cuSten->boundaryTop[pt_cuSten->numTiles - 1] = &dataInput[(pt_cuSten->nyTile * (pt_cuSten->numTiles - 1) - pt_cuSten->numStenTop) * pt_cuSten->nx];
			pt_cuSten->boundaryBottom[pt_cuSten->numTiles - 1] = &dataInput[0];

			break;
	}

	// Number of points in top boundary data
	pt_cuSten->numBoundaryTop = pt_cuSten->numStenTop * pt_cuSten->nx;

	// Number of points in bottom boundary data
	pt_cuSten->numBoundaryBottom = pt_cuSten->numStenBottom * pt_cuSten->nx;

}
// ---------------------------------------------------------------------
// Swap pointers
// ---------------------------------------------------------------------

/*! \fun void cuStenSwap2DYp
    \brief Function to swap pointers necessary for timestepping
    \param pt_cuSten Pointer to cuSten type provided by user
	\param dataInput Pointer to data input to the on the next compute
*/

template <typename elemType>
void cuStenSwap2DYp(
	cuSten_t<elemType>* pt_cuSten,
	elemType* dataInput
) 
{
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{	
		// Swap the input and output data
		std::swap(pt_cuSten->dataInput[tile], pt_cuSten->dataOutput[tile]);

		// Update the boundary data
		switch(pt_cuSten->numTiles)
		{
			// One tile only requires single top and bottom to be set
			case 1:
				pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[0] = &dataInput[0]; 

				break;

			// Two tiles requires a special case of only setting two tiles
			case 2:
				pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[0] = &dataInput[pt_cuSten->nyTile * pt_cuSten->nx];

				pt_cuSten->boundaryTop[1] = &dataInput[(pt_cuSten->nyTile - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[1] = &dataInput[0];

				break;

			// Default case has interiors, so set the top tile, then loop over interior, then set the bottom tile
			default:
				pt_cuSten->boundaryTop[0] = &dataInput[(pt_cuSten->ny - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[0] = &dataInput[pt_cuSten->nyTile * pt_cuSten->nx];

				for (int tile = 1; tile < pt_cuSten->numTiles - 1; tile++)
				{
					pt_cuSten->boundaryTop[tile] = &dataInput[(pt_cuSten->nyTile * tile - pt_cuSten->numStenTop) * pt_cuSten->nx];
					pt_cuSten->boundaryBottom[tile] = &dataInput[(pt_cuSten->nyTile * (tile + 1)) * pt_cuSten->nx];
				}

				pt_cuSten->boundaryTop[pt_cuSten->numTiles - 1] = &dataInput[(pt_cuSten->nyTile * (pt_cuSten->numTiles - 1) - pt_cuSten->numStenTop) * pt_cuSten->nx];
				pt_cuSten->boundaryBottom[pt_cuSten->numTiles - 1] = &dataInput[0];

				break;
		}
	}
}

// ---------------------------------------------------------------------
// Function to destroy the struct
// ---------------------------------------------------------------------

/*! \fun void cuStenDestroy2DYp
    \brief Function to destroy data associated with cuSten_t
    \param pt_cuSten Pointer to cuSten type provided by user
*/

template <typename elemType>
void cuStenDestroy2DYp(
	cuSten_t<elemType>* pt_cuSten
) 
{
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);	


	// Destroy the streams
	for (int st = 0; st < pt_cuSten->numStreams; st++)
	{
		hipStreamDestroy(pt_cuSten->streams[st]);
		sprintf(msgStringBuffer, "Destroying stream %d on GPU %d", st, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);	
	}

	// Free the main memory
	free(pt_cuSten->streams);

	// // Create the events
	for (int ev = 0; ev < 2; ev++)
	{
		hipEventDestroy(pt_cuSten->events[ev]);
		sprintf(msgStringBuffer, "Destroying event %d on GPU %d", ev, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);
	}

	// Free the main memory
	free(pt_cuSten->events);

	// Free the pointers for each input tile
	free(pt_cuSten->dataInput);

	// Free the pointers for each output tile
	free(pt_cuSten->dataOutput);

	// Free the top boundary tile pointers
	free(pt_cuSten->boundaryTop);

	// Free the bottom boundary tile pointers
	free(pt_cuSten->boundaryBottom);
}

// ---------------------------------------------------------------------
// Explicit instantiation
// ---------------------------------------------------------------------

template
void cuStenCreate2DYp<double>(
	cuSten_t<double>*,
	int,
	int,
	int,
	int,
	int,
	int,
	double*,
	double*,
	double*,
	int,
	int,
	int
);

template
void cuStenSwap2DYp<double>(
	cuSten_t<double>*,
	double*
);

template
void cuStenDestroy2DYp<double>(
	cuSten_t<double>*
);

template
void cuStenCreate2DYp<float>(
	cuSten_t<float>*,
	int,
	int,
	int,
	int,
	int,
	int,
	float*,
	float*,
	float*,
	int,
	int,
	int
);

template
void cuStenSwap2DYp<float>(
	cuSten_t<float>*,
	float*
);

template
void cuStenDestroy2DYp<float>(
	cuSten_t<float>*
);

// ---------------------------------------------------------------------
// End of file
// ---------------------------------------------------------------------
