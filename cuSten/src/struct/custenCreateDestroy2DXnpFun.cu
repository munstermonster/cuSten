// Andrew Gloster
// May 2018

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.

/*! \file custenCreateDestroy2DXnpFun.cu
    Functions to create and destroy the cuSten_t that is used to give input to the compute kernels. 
    2D x direction, non-periodic, user function
*/

// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <iostream>

// ---------------------------------------------------------------------
// Custom libraries and headers
// ---------------------------------------------------------------------

#include "cuSten_struct_type.h"
#include "cuSten_struct_functions.h"
#include "../util/util.h"

// ---------------------------------------------------------------------
// Function to create the struct
// ---------------------------------------------------------------------

/*! \fun void cuStenCreate2DXnpFun
    \brief Function to set up cuSten_t
    \param pt_cuSten Pointer to cuSten type provided by user
	\param numTiles Number of tiles to divide the data into
	\param nx Total number of points in the x direction 
	\param ny Total number of points in the y direction 
	\param BLOCK_X Size of thread block in the x direction
	\param BLOCK_Y Size of thread block in the y direction
    \param dataOutput Pointer to data output by the function
	\param dataInput Pointer to data input to the function    
	\param numSten Total number of points in the stencil
	\param numStenLeft Number of points on the left side of the stencil
	\param numStenRight Number of points on the right side of the stencil
	\param Number of coefficients used by the user in their function
	\param Pointer to user function
*/

template <typename elemType>
void cuStenCreate2DXnpFun(
	cuSten_t<elemType>* pt_cuSten,
	int deviceNum,
	int numTiles,
	int nx,
	int ny,
	int BLOCK_X,
	int BLOCK_Y,
	elemType* dateOutput,
	elemType* dateInput,
	elemType* coe,
	int numSten,
	int numStenLeft,
	int numStenRight,
	int numCoe,
	elemType* func
) 
{
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set the device number associated with the struct
  	pt_cuSten->deviceNum = deviceNum;

  	// Set the number of streams
  	pt_cuSten->numStreams = 3;

  	// Set the number of tiles
  	pt_cuSten->numTiles = numTiles;

  	// Set the number points in x on the device
  	pt_cuSten->nx = nx;

  	// Set the number points in y on the device
  	pt_cuSten->ny = ny;

  	// Number of threads in x on the device
	pt_cuSten->BLOCK_X = BLOCK_X;

  	// Number of threads in y on the device
	pt_cuSten->BLOCK_Y = BLOCK_Y;

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);	

	// Create memeory for the streams
	pt_cuSten->streams = (hipStream_t*)malloc(pt_cuSten->numStreams * sizeof(hipStream_t*));

	// Create the streams
	for (int st = 0; st < pt_cuSten->numStreams; st++)
	{
		hipStreamCreate(&pt_cuSten->streams[st]);
		sprintf(msgStringBuffer, "Creating stream %d on GPU %d", st, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);	
	}

	// Create memeory for the events
	pt_cuSten->events = (hipEvent_t*)malloc(2 * sizeof(hipEvent_t*));

	// Create the events
	for (int ev = 0; ev < 2; ev++)
	{
		hipEventCreate(&pt_cuSten->events[ev]);
		sprintf(msgStringBuffer, "Creating event %d on GPU %d", ev, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);
	}

	// Set number of points in the stencil
	pt_cuSten->numSten = numSten;

	// Set number of points to the left in the stencil
	pt_cuSten->numStenLeft = numStenLeft;

	// Set number of points to the right in the stencil
	pt_cuSten->numStenRight = numStenRight;

	// Set the device coefficients pointer
	pt_cuSten->coe = coe;

	// Set number of coefficients
	pt_cuSten->numCoe = numCoe;

	// Local memory grid sizes
	pt_cuSten->nxLocal = pt_cuSten->BLOCK_X + pt_cuSten->numStenLeft + pt_cuSten->numStenRight;
	pt_cuSten->nyLocal = pt_cuSten->BLOCK_Y;

	// Set the amount of shared memory required
	pt_cuSten->mem_shared = pt_cuSten->nxLocal * pt_cuSten->nyLocal * sizeof(elemType) + numCoe * sizeof(elemType);

	// Find number of points per tile
	pt_cuSten->nx = pt_cuSten->nx;

	// Find number of points per tile
	pt_cuSten->nyTile = pt_cuSten->ny / pt_cuSten->numTiles;	

	// Set the grid up
    pt_cuSten->xGrid = (pt_cuSten->nx % pt_cuSten->BLOCK_X == 0) ? (pt_cuSten->nx / pt_cuSten->BLOCK_X) : (pt_cuSten->nx / pt_cuSten->BLOCK_X + 1);
    pt_cuSten->yGrid = (pt_cuSten->nyTile % pt_cuSten->BLOCK_Y == 0) ? (pt_cuSten->nyTile / pt_cuSten->BLOCK_Y) : (pt_cuSten->nyTile / pt_cuSten->BLOCK_Y + 1);

	// Allocate the pointers for each input tile
	pt_cuSten->dataInput = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// Allocate the pointers for each output tile
	pt_cuSten->dataOutput = (elemType**)malloc(pt_cuSten->numTiles * sizeof(elemType));

	// Tile offset index
	int offset = pt_cuSten->nx * pt_cuSten->nyTile;

	// // Match the pointers to the data
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{	
		// Set the input data
		pt_cuSten->dataInput[tile] = &dateInput[tile * offset];

		// Set the output data
		pt_cuSten->dataOutput[tile] = &dateOutput[tile * offset];
	}

	// Set the function
	pt_cuSten->devFunc = func;

}

// ---------------------------------------------------------------------
// Swap pointers
// ---------------------------------------------------------------------

/*! \fun void cuStenSwap2DXnpFun
    \brief Function to swap pointers necessary for timestepping
    \param pt_cuSten Pointer to cuSten type provided by user
	\param dataInput Pointer to data input to the on the next compute
*/

template <typename elemType>
void cuStenSwap2DXnpFun(
	cuSten_t<elemType>* pt_cuSten,
	elemType* dataInput
) 
{
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{	
		// Swap the input and output data
		std::swap(pt_cuSten->dataInput[tile], pt_cuSten->dataOutput[tile]);
	}
}

// ---------------------------------------------------------------------
// Function to destroy the struct
// ---------------------------------------------------------------------

/*! \fun void cuStenDestroy2DXnpFun
    \brief Function to destroy data associated with cuSten_t
    \param pt_cuSten Pointer to cuSten type provided by user
*/

template <typename elemType>
void cuStenDestroy2DXnpFun(
	cuSten_t<elemType>* pt_cuSten
) 
{
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);	

	// Destroy the streams
	for (int st = 0; st < pt_cuSten->numStreams; st++)
	{
		hipStreamDestroy(pt_cuSten->streams[st]);
		sprintf(msgStringBuffer, "Destroying stream %d on GPU %d", st, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);	
	}

	// Free the main memory
	free(pt_cuSten->streams);

	// // Create the events
	for (int ev = 0; ev < 2; ev++)
	{
		hipEventDestroy(pt_cuSten->events[ev]);
		sprintf(msgStringBuffer, "Destroying event %d on GPU %d", ev, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);
	}

	// Free the main memory
	free(pt_cuSten->events);

	// Free the pointers for each input tile
	free(pt_cuSten->dataInput);

	// Free the pointers for each output tile
	free(pt_cuSten->dataOutput);
}

// ---------------------------------------------------------------------
// Explicit instantiation
// ---------------------------------------------------------------------

template
void cuStenCreate2DXnpFun<double>(
	cuSten_t<double>*,
	int,
	int,
	int,
	int,
	int,
	int,
	double*,
	double*,
	double*,
	int,
	int,
	int,
	int,
	double*
);

template
void cuStenSwap2DXnpFun<double>(
	cuSten_t<double>*,
	double* dataInput
);

template
void cuStenDestroy2DXnpFun<double>(
	cuSten_t<double>*
);

template
void cuStenCreate2DXnpFun<float>(
	cuSten_t<float>*,
	int,
	int,
	int,
	int,
	int,
	int,
	float*,
	float*,
	float*,
	int,
	int,
	int,
	int,
	float*
);

template
void cuStenSwap2DXnpFun<float>(
	cuSten_t<float>*,
	float* dataInput
);

template
void cuStenDestroy2DXnpFun<float>(
	cuSten_t<float>*
);

// ---------------------------------------------------------------------
// End of file
// ---------------------------------------------------------------------
