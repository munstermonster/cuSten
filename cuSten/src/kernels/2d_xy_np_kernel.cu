#include "hip/hip_runtime.h"
// Andrew Gloster
// January 2019

//   Copyright 2019 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.

/*! \file 2d_xy_np_kernel.cu
    Kernel to apply a xy direction stencil on a 2D grid - non periodic
*/

// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <iostream>
#include <cstdio>

// ---------------------------------------------------------------------
// Custom libraries and headers
// ---------------------------------------------------------------------

#include "../util/util.h"
#include "../struct/cuSten_struct_type.h"

// ---------------------------------------------------------------------
//  Kernel Definition
// ---------------------------------------------------------------------

/*! \fun __global__ void kernel2DXYnp
    \brief Device function to apply the stencil to the data and output the answer.
    \param dataOutput Pointer to data output by the function
	\param dataInput Pointer to data input to the function
	\param boundaryTop Pointer to data in the top boundary of the current tile
	\param boundaryBottom Pointer to data in the bottom boundary of the current tile
	\param weights Pointer to coefficients to be used in stencil
	\param numSten Total number of points in the stencil
	\param numStenHoriz Number of points in a horizontal stencil
	\param numStenLeft Number of points on the left side of the stencil
	\param numStenRight Number of points on the right side of the stencil
	\param numStenVert Number of points in a vertical stencil
	\param numStenTop Number of points on top of stencil
	\param numStenBottom Number of points on bottom of stencil
	\param nxLocal Number of points in sharded memory in the x direction
	\param nyLocal Number of points in sharded memory in the y direction
	\param BLOCK_X Size of thread block in the x direction
	\param BLOCK_Y Size of thread block in the y direction
	\param nx Total number of points in the x direction
	\param nyTile Number of y direction points on tile
	\param tileTop Check if the current tile is at the top of the domain
	\param tileBottom Check if the current tile is at the bottom of the domain
*/

template <typename elemType>
__global__ void kernel2DXYnp
(
	elemType* dataOutput,  				
	elemType* dataInput,					
	elemType* boundaryTop, 				
	elemType* boundaryBottom,				
	const elemType* weights,       		
	const int numSten,
	const int numStenHoriz,				
	const int numStenLeft,				
	const int numStenRight,				
	const int numStenVert,				
	const int numStenTop,				
	const int numStenBottom,			
	const int nxLocal,					
	const int nyLocal,					
	const int BLOCK_X, 					
	const int BLOCK_Y,					
	const int nx,					
	const int nyTile,					
	const int tileTop,					
	const int tileBottom				
)
{	
	// -----------------------------	
	// Allocate the shared memory
	// -----------------------------

	extern __shared__ int memory[];

	elemType* arrayLocal = (elemType*)&memory;
	elemType* weigthsLocal = (elemType*)&arrayLocal[nxLocal * nyLocal];

	// Move the weigths into shared memory
	#pragma unroll
	for (int k = 0; k < numSten; k++)
	{
		weigthsLocal[k] = weights[k];
	}

	// -----------------------------
	// Set the indexing
	// -----------------------------

	// True matrix index
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

	// Local matrix index
	int localIdx = threadIdx.x + numStenLeft;
	int localIdy = threadIdx.y + numStenTop;

	// Local sum variable
	elemType sum = 0.0;

	// Set index for summing stencil
	int stenSet;

	// Set temporary index for looping
	int temp;

	// Use to loop over indexing in the weighsLocal
	int weight = 0;

	// -----------------------------
	// We divide the domain in 9 - 4x Corners, 4x Edges, 1x Interior
	// -----------------------------

	// -----------------------------
	// (0, 0) - Top Left
	// -----------------------------

	if (blockIdx.x == 0 && blockIdx.y == 0)
	{
		// ----------
		// Copy
		// ----------
	
		// Interior
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		// Right
		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
		}

		// Bottom
		if (threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
		}

		// Bottom Right
		if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx + BLOCK_X)];
		}

		// Top
		if (tileTop != 1)
		{	
			// Top
			if (threadIdx.y < numStenTop)
			{
				arrayLocal[threadIdx.y * nxLocal + localIdx] = boundaryTop[threadIdx.y * nx + globalIdx];
			}

			// Top Right
			if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
			{
				arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = boundaryTop[threadIdx.y * nx + (globalIdx + BLOCK_X)];
			}
		}

		// Ensure copying completed
		__syncthreads();

		// ----------
		// Compute
		// ----------

		stenSet = threadIdx.y * nxLocal + threadIdx.x;
		weight = 0;

		for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
		{
			temp = j * nxLocal;

			for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
			{
				sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

				weight++;
			} 
		}

		// Ensure the compute is complete
		__syncthreads();

		// ----------
		// Copy back 
		// ----------

		if (tileTop == 1)
		{
			if (threadIdx.x >= numStenLeft && threadIdx.y >= numStenTop)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}
		else
		{
			if (threadIdx.x >= numStenLeft)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}
		
	}

	// -----------------------------
	// (nx / BLOCK_X - 1, 0) - Top Right
	// -----------------------------

	else if (blockIdx.x == nx / BLOCK_X - 1 && blockIdx.y == 0)
	{
		// ----------
		// Copy
		// ----------

		// Interior
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		// Left 
		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
		}

		// Bottom
		if (threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
		}

		// Bottom Left
		if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx - numStenLeft)];
		}

		// Top
		if (tileTop != 1)
		{	
			// Top
			if (threadIdx.y < numStenTop)
			{
				arrayLocal[threadIdx.y * nxLocal + localIdx] = boundaryTop[threadIdx.y * nx + globalIdx];
			}

			// Top Left
			if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
			{
				arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = boundaryTop[threadIdx.y * nx + (globalIdx - numStenLeft)];
			}
		}

		// Ensure copying completed
		__syncthreads();

		// ----------
		// Compute
		// ----------

		stenSet = threadIdx.y * nxLocal + threadIdx.x;
		weight = 0;

		for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
		{
			temp = j * nxLocal;

			for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
			{
				sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

				weight++;
			} 
		}

		// Ensure the compute is complete
		__syncthreads();

		// ----------
		// Copy back 
		// ----------

		if (tileTop == 1)
		{
			if (threadIdx.x < BLOCK_X - numStenRight && threadIdx.y >= numStenTop)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}
		else
		{
			if (threadIdx.x < BLOCK_X - numStenRight)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}

	}

	// -----------------------------
	// (0, nyTile / BLOCK_Y - 1) - Bottom Left
	// -----------------------------

	else if (blockIdx.x == 0 && blockIdx.y == nyTile / BLOCK_Y - 1)
	{
		// ----------
		// Copy
		// ----------

		// Interior
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		// Right
		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
		}

		// Top
		if (threadIdx.y < numStenTop )
		{
			arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
		}

		// Top Right
		if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx + BLOCK_X)];
		}

		if (tileBottom != 1)
		{
			// Bottom
			if (threadIdx.y < numStenBottom)
			{
				arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = boundaryBottom[threadIdx.y * nx + globalIdx];
			}

			// Bottom Right
			if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
			{
				arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] =  boundaryBottom[threadIdx.y * nx + (globalIdx + BLOCK_X)];
			}
		}

		// Ensure the copy is complete
		__syncthreads();

		// ----------
		// Compute
		// ----------

		stenSet = threadIdx.y * nxLocal + threadIdx.x;
		weight = 0;

		for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
		{
			temp = j * nxLocal;

			for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
			{
				sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

				weight++;
			} 
		}

		// Ensure the compute is complete
		__syncthreads();

		// ----------
		// Copy back 
		// ----------

		if (tileBottom == 1)
		{
			if (threadIdx.x >= numStenLeft && threadIdx.y < BLOCK_Y - numStenBottom)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}
		else
		{
			if (threadIdx.x >= numStenLeft)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}
	}

	// -----------------------------
	// (0, nyTile / BLOCK_Y - 1) - Bottom Right
	// -----------------------------

	else if (blockIdx.x == nx / BLOCK_X - 1 && blockIdx.y == nyTile / BLOCK_Y - 1)
	{
		// ----------
		// Copy
		// ----------

		// Interior
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		// Left 
		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
		}

		// Top
		if (threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
		}

		// Top Left
		if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx - numStenLeft)];
		}

		if (tileBottom != 1)
		{
			// Bottom
			if (threadIdx.y < numStenBottom)
			{
				arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = boundaryBottom[threadIdx.y * nx + globalIdx];
			}

			// Bottom Left
			if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
			{
				arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = boundaryBottom[threadIdx.y * nx + (globalIdx - numStenLeft)];
			}
		}

		// Ensure copying completed
		__syncthreads();

		// ----------
		// Compute
		// ----------

		stenSet = threadIdx.y * nxLocal + threadIdx.x;
		weight = 0;

		for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
		{
			temp = j * nxLocal;

			for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
			{
				sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

				weight++;
			} 
		}

		// Ensure the compute is complete
		__syncthreads();

		// ----------
		// Copy back 
		// ----------

		if (tileBottom == 1)
		{
			if (threadIdx.x < BLOCK_X - numStenRight && threadIdx.y < BLOCK_Y - numStenBottom)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}
		else
		{
			if (threadIdx.x < BLOCK_X - numStenRight)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}
	}

	// -----------------------------
	// (_, 0) - Top
	// -----------------------------

	else if (blockIdx.y == 0)
	{
		// ----------
		// Copy
		// ----------
	
		// Interior
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		// Left 
		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
		}
			
		// Right
		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
		}

		// Bottom
		if (threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
		}

		// Bottom Right
		if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx + BLOCK_X)];
		}

		// Bottom Left
		if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx - numStenLeft)];
		}

		if (tileTop != 1)
		{
			// Top
			if (threadIdx.y < numStenTop)
			{
				arrayLocal[threadIdx.y * nxLocal + localIdx] = boundaryTop[threadIdx.y * nx + globalIdx];
			}

			// Top Left
			if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
			{
				arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = boundaryTop[threadIdx.y * nx + (globalIdx - numStenLeft)];
			}

			// Top Right
			if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
			{
				arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = boundaryTop[threadIdx.y * nx + (globalIdx + BLOCK_X)];
			}
		}

		// Ensure copying completed
		__syncthreads();

		// ----------
		// Compute
		// ----------

		stenSet = threadIdx.y * nxLocal + threadIdx.x;
		weight = 0;

		for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
		{
			temp = j * nxLocal;

			for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
			{
				sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

				weight++;
			} 
		}

		// Ensure the compute is complete
		__syncthreads();

		// ----------
		// Copy back 
		// ----------

		if (tileTop == 1)
		{
			if (threadIdx.y >= numStenTop)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}
		else
		{
			dataOutput[globalIdy * nx + globalIdx] = sum;
		}
	}

	// -----------------------------
	// (_, nyTile / BLOCK_Y - 1) - Bottom
	// -----------------------------

	else if (blockIdx.y == nyTile / BLOCK_Y - 1)
	{
		// ----------
		// Copy
		// ----------
	
		// Interior
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		// Left 
		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
		}
			
		// Right
		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
		}

		// Top
		if (threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
		}

		// Top Left
		if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx - numStenLeft)];
		}

		// Top Right
		if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx + BLOCK_X)];
		}

		if (tileBottom != 1)
		{
			// Bottom
			if (threadIdx.y < numStenBottom)
			{
				arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = boundaryBottom[threadIdx.y * nx + globalIdx];
			}

			// Bottom Left
			if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
			{
				arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = boundaryBottom[threadIdx.y * nx + (globalIdx - numStenLeft)];
			}

			// Bottom Right
			if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
			{
				arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] =  boundaryBottom[threadIdx.y * nx + (globalIdx + BLOCK_X)];
			}
		}	

		// Ensure copying completed
		__syncthreads();
		
		// ----------
		// Compute
		// ----------

		stenSet = threadIdx.y * nxLocal + threadIdx.x;
		weight = 0;

		for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
		{
			temp = j * nxLocal;

			for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
			{
				sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

				weight++;
			} 
		}

		// Ensure the compute is complete
		__syncthreads();

		// ----------
		// Copy back 
		// ----------

		if (tileBottom == 1)
		{
			if (threadIdx.y < BLOCK_Y - numStenBottom)
			{
				dataOutput[globalIdy * nx + globalIdx] = sum;
			}
		}
		else
		{
			dataOutput[globalIdy * nx + globalIdx] = sum;
		}	
	}

	// -----------------------------
	// (0, _) - Left
	// -----------------------------

	else if (blockIdx.x == 0)
	{
		// ----------
		// Copy
		// ----------
	
		// Interior
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		// Right
		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
		}

		// Top
		if (threadIdx.y < numStenTop )
		{
			arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
		}

		// Top Right
		if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx + BLOCK_X)];
		}

		// Bottom
		if (threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
		}

		// Bottom Right
		if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx + BLOCK_X)];
		}

		// Ensure copying completed
		__syncthreads();
		
		// ----------
		// Compute
		// ----------

		stenSet = threadIdx.y * nxLocal + threadIdx.x;
		weight = 0;

		for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
		{
			temp = j * nxLocal;

			for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
			{
				sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

				weight++;
			} 
		}

		// Ensure the compute is complete
		__syncthreads();

		// ----------
		// Copy back 
		// ----------

		if (threadIdx.x >= numStenLeft)
		{
			dataOutput[globalIdy * nx + globalIdx] = sum;
		}	
	}

	// -----------------------------
	// (nx / BLOCK_X - 1, _) - Right
	// -----------------------------

	else if (blockIdx.x == nx / BLOCK_X - 1)
	{
		// ----------
		// Copy
		// ----------
	
		// Interior
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		// Left 
		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
		}

		// Top
		if (threadIdx.y < numStenTop )
		{
			arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
		}

		// Top Left
		if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx - numStenLeft)];
		}

		// Bottom
		if (threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
		}

		// Bottom Left
		if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx - numStenLeft)];
		}

		// Ensure copying completed
		__syncthreads();
		
		// ----------
		// Compute
		// ----------

		stenSet = threadIdx.y * nxLocal + threadIdx.x;
		weight = 0;

		for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
		{
			temp = j * nxLocal;

			for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
			{
				sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

				weight++;
			} 
		}

		// Ensure the compute is complete
		__syncthreads();

		// ----------
		// Copy back 
		// ----------

		if (threadIdx.x < BLOCK_X - numStenLeft)
		{
			dataOutput[globalIdy * nx + globalIdx] = sum;
		}
	}

	// -----------------------------
	// Interior
	// -----------------------------

	else
	{
		// ----------
		// Copy
		// ----------
	
		// Interior
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		// Left 
		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
		}

		// Right
		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
		}

		// Top
		if (threadIdx.y < numStenTop )
		{
			arrayLocal[threadIdx.y * nxLocal + localIdx] = dataInput[(globalIdy - numStenTop) * nx + globalIdx];
		}

		// Top Left
		if (threadIdx.x < numStenLeft && threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + threadIdx.x] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx - numStenLeft)];
		}

		// Top Right
		if (threadIdx.x < numStenRight && threadIdx.y < numStenTop)
		{
			arrayLocal[threadIdx.y * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy - numStenTop) * nx + (globalIdx + BLOCK_X)];
		}

		// Bottom
		if (threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = dataInput[(globalIdy + BLOCK_Y) * nx + globalIdx];
		}

		// Bottom Left
		if (threadIdx.x < numStenLeft && threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + threadIdx.x] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx - numStenLeft)];
		}

		// Bottom Right
		if (threadIdx.x < numStenRight && threadIdx.y < numStenBottom)
		{
			arrayLocal[(localIdy + BLOCK_Y) * nxLocal + (localIdx + BLOCK_X)] = dataInput[(globalIdy + BLOCK_Y) * nx + (globalIdx + BLOCK_X)];
		}

		// Ensure copying completed
		__syncthreads();
		
		// ----------
		// Compute
		// ----------

		stenSet = threadIdx.y * nxLocal + threadIdx.x;
		weight = 0;

		for (int j = 0; j < numStenVert; j++) // Allow for the point we're actually at
		{
			temp = j * nxLocal;

			for (int i = 0; i < numStenHoriz; i++) // Allow for the point we're actually at
			{
				sum += weigthsLocal[weight] * arrayLocal[stenSet + temp + i];

				weight++;
			} 
		}

		// Ensure the compute is complete
		__syncthreads();

		// ----------
		// Copy back 
		// ----------

		dataOutput[globalIdy * nx + globalIdx] = sum;
	}
}

// ---------------------------------------------------------------------
// Function to compute kernel
// ---------------------------------------------------------------------

/*! \fun void cuStenCompute2DXYnp
    \brief Function called by user to compute the stencil.
    \param pt_cuSten Pointer to cuSten data type which contains all the necessary input
	\param offload Set to HOST to move data back to CPU or DEVICE to keep on the GPU
*/

template <typename elemType>
void cuStenCompute2DXYnp
(
	cuSten_t<elemType>* pt_cuSten,
	bool offload
)
{	
	// 	Buffer used for error checking
	char msgStringBuffer[1024];

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);

	dim3 blockDim(pt_cuSten->BLOCK_X, pt_cuSten->BLOCK_Y);
	dim3 gridDim(pt_cuSten->xGrid, pt_cuSten->yGrid);

	// Load the weights
	hipMemPrefetchAsync(pt_cuSten->weights, pt_cuSten->numSten * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);

	// Ensure the current stream is free
	hipStreamSynchronize(pt_cuSten->streams[1]);

	// Prefetch the tile data
	hipMemPrefetchAsync(pt_cuSten->dataInput[0], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
	hipMemPrefetchAsync(pt_cuSten->dataOutput[0], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);

	// Prefetch the boundary data
	hipMemPrefetchAsync(pt_cuSten->boundaryTop[0], pt_cuSten->numBoundaryTop * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
	hipMemPrefetchAsync(pt_cuSten->boundaryBottom[0], pt_cuSten->numBoundaryTop * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);

	// Record the event
	hipEventRecord(pt_cuSten->events[0], pt_cuSten->streams[1]);

	// Temporary stream and event used for permuting
	hipStream_t ts;
	hipEvent_t te;

	// Tile positions
	int tileTop;
	int tileBottom;

	// Loop over the tiles
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{
		// Set the variables that describe the current tile position
		if (pt_cuSten->numTiles == 1)
		{
			tileTop = 1;
			tileBottom = 1;
		}
		else
		{
			if (tile == 0)
			{
				tileTop = 1;
				tileBottom = 0;
			}
			else if (tile == pt_cuSten->numTiles - 1)
			{
				tileTop = 0;
				tileBottom = 1;
			}
			else
			{
				tileTop = 0;
				tileBottom = 0;
			}
		}
		// Synchronise the events to ensure computation overlaps
		hipEventSynchronize(pt_cuSten->events[0]);

		// Preform the computation on the current tile
		kernel2DXYnp<<<gridDim, blockDim, pt_cuSten->mem_shared, pt_cuSten->streams[0]>>>(
			pt_cuSten->dataOutput[tile], 

			pt_cuSten->dataInput[tile], 

			pt_cuSten->boundaryTop[tile], 
			pt_cuSten->boundaryBottom[tile], 

			pt_cuSten->weights, 

			pt_cuSten->numSten,

			pt_cuSten->numStenHoriz,
			pt_cuSten->numStenLeft,
			pt_cuSten->numStenRight,

			pt_cuSten->numStenVert,
			pt_cuSten->numStenTop, 
			pt_cuSten->numStenBottom,

			pt_cuSten->nxLocal, 
			pt_cuSten->nyLocal,

			pt_cuSten->BLOCK_X, 
			pt_cuSten->BLOCK_Y, 

			pt_cuSten->nx, 
			pt_cuSten->nyTile,

			tileTop,
			tileBottom
		);

		sprintf(msgStringBuffer, "Error computing tile %d on GPU %d", tile, pt_cuSten->deviceNum);
		checkError(msgStringBuffer);	

		// Offload should the user want to
		if (offload == 1)
		{
			hipMemPrefetchAsync(pt_cuSten->dataOutput[tile], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), hipCpuDeviceId, pt_cuSten->streams[0]);
	    	hipMemPrefetchAsync(pt_cuSten->dataInput[tile], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), hipCpuDeviceId, pt_cuSten->streams[0]);
		}

		// Load the next set of data
    	if (tile < pt_cuSten->numTiles - 1)
    	{
			// Ensure the current stream is free
    		hipStreamSynchronize(pt_cuSten->streams[1]);

    		// Prefetch the tiles
			hipMemPrefetchAsync(pt_cuSten->dataOutput[tile + 1], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
		 	hipMemPrefetchAsync(pt_cuSten->dataInput[tile + 1], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);

		 	// Prefetch the next boundaries
		 	hipMemPrefetchAsync(pt_cuSten->boundaryTop[tile + 1], pt_cuSten->numBoundaryTop * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
			hipMemPrefetchAsync(pt_cuSten->boundaryBottom[tile + 1], pt_cuSten->numBoundaryBottom * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);

			// Record the event
			hipEventRecord(pt_cuSten->events[1], pt_cuSten->streams[1]);
    	}

    	// Permute streams
    	for (int i = 0; i < pt_cuSten->numStreams - 1; i++)
    	{
    		ts = pt_cuSten->streams[i];
    		pt_cuSten->streams[i] = pt_cuSten->streams[i + 1];	
    		pt_cuSten->streams[i + 1] = ts;    			
    	}

    	// Permute events
		te = pt_cuSten->events[0]; pt_cuSten->events[0] = pt_cuSten->events[1]; pt_cuSten->events[1] = te; 
    }
}

// ---------------------------------------------------------------------
// Explicit instantiation
// ---------------------------------------------------------------------

template
__global__ void kernel2DXYnp<double>
(
	double*,  				
	double*,					
	double*, 				
	double*,				
	const double*,       		
	const int,
	const int,				
	const int,				
	const int,			
	const int,			
	const int,				
	const int,			
	const int,					
	const int,					
	const int, 					
	const int,					
	const int,					
	const int,					
	const int,					
	const int				
);

template
void cuStenCompute2DXYnp<double>
(
	cuSten_t<double>*,
	bool
);

template
__global__ void kernel2DXYnp<float>
(
	float*,  				
	float*,					
	float*, 				
	float*,				
	const float*,       		
	const int,
	const int,				
	const int,				
	const int,			
	const int,			
	const int,				
	const int,			
	const int,					
	const int,					
	const int, 					
	const int,					
	const int,					
	const int,					
	const int,					
	const int				
);

template
void cuStenCompute2DXYnp<float>
(
	cuSten_t<float>*,
	bool
);

// ---------------------------------------------------------------------
// End of file
// ---------------------------------------------------------------------
