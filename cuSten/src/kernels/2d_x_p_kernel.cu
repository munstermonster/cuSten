#include "hip/hip_runtime.h"
// Andrew Gloster
// May 2018
// Kernel to apply an x direction stencil on a 2D grid - non periodic

//   Copyright 2018 Andrew Gloster

//   Licensed under the Apache License, Version 2.0 (the "License");
//   you may not use this file except in compliance with the License.
//   You may obtain a copy of the License at

//       http://www.apache.org/licenses/LICENSE-2.0

//   Unless required by applicable law or agreed to in writing, software
//   distributed under the License is distributed on an "AS IS" BASIS,
//   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//   See the License for the specific language governing permissions and
//   limitations under the License.

/*! \file 2d_x_p_kernel.cu
    Functions to apply a periodic stencil to a 2D domain, x-direction only.
*/

// ---------------------------------------------------------------------
//  Standard Libraries and Headers
// ---------------------------------------------------------------------

#include <iostream>
#include <cstdio>

// ---------------------------------------------------------------------
// Custom libraries and headers
// ---------------------------------------------------------------------

#include "../util/util.h"
#include "../struct/cuSten_struct_type.h"

// ---------------------------------------------------------------------
//  Kernel Definition
// ---------------------------------------------------------------------

/*! \fun static __global__ void kernel2DXnp
    \brief Device function to apply the stencil to the data and output the answer.
    \param dataOutput Pointer to data output by the function
	\param dataInput Pointer to data input to the function
	\param weights Pointer to coefficients to be used in stencil
	\param numSten Total number of points in the stencil
	\param numStenLeft Number of points on the left side of the stencil
	\param numStenRight Number of points on the right side of the stencil
	\param nxLocal Number of points in sharded memory in the x direction
	\param nyLocal Number of points in sharded memory in the y direction
	\param BLOCK_X Size of thread block in the x direction
	\param nx Total number of points in the x direction
*/

template <typename elemType>
__global__ void kernel2DXp
(
	elemType* dataOutput,  					
	elemType* dataInput,					
	const elemType* weights,       		
	const int numSten,					
	const int numStenLeft,				
	const int numStenRight,				
	const int nxLocal,					
	const int nyLocal,					
	const int BLOCK_X,					
	const int nx
)
{	
	// -----------------------------	
	// Allocate the shared memory
	// -----------------------------

	extern __shared__ int memory[];
	
	elemType* arrayLocal = (elemType*)&memory;
	elemType* weigthsLocal = (elemType*)&arrayLocal[nxLocal * nyLocal];

	// Move the weigths into shared memory
	#pragma unroll
	for (int k = 0; k < numSten; k++)
	{
		weigthsLocal[k] = weights[k];
	}

	// -----------------------------
	// Set the indexing
	// -----------------------------

	// True matrix index
	int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
	int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

	// Local matrix index
	int localIdx = threadIdx.x + numStenLeft;
	int localIdy = threadIdx.y;

	// Local sum variable
	elemType sum = 0.0;

	// Set index for summing stencil
	int stenSet;

	// -----------------------------
	// Set interior
	// -----------------------------

	arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

	// -----------------------------
	// Set x boundaries
	// -----------------------------

	// If block is in the interior
	if (blockIdx.x != 0 && blockIdx.x != nx / BLOCK_X - 1)
	{

		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
		}

		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
		}
	}

	// If block is on the left boundary
	if (blockIdx.x == 0)
	{
		arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (nx - numStenLeft + threadIdx.x)];
		}

		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
		}
	}

	// Set the right boundary blocks
	if (blockIdx.x == nx / BLOCK_X - 1)
	{
		arrayLocal[localIdy * nxLocal + threadIdx.x + numStenLeft] = dataInput[globalIdy * nx + globalIdx];

		if (threadIdx.x < numStenLeft)
		{
			arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
		}

		if (threadIdx.x < numStenRight)
		{
			arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + threadIdx.x];
		}
	}

	// -----------------------------
	// Compute the stencil
	// -----------------------------

	__syncthreads();

	stenSet = localIdy * nxLocal + threadIdx.x;

	#pragma unroll
	for (int k = 0; k < numSten; k++)
	{
		sum += weigthsLocal[k] * arrayLocal[stenSet + k];
	}

	__syncthreads();

	// -----------------------------
	// Copy back to global
	// -----------------------------

	dataOutput[globalIdy * nx + globalIdx] = sum;
}

// ---------------------------------------------------------------------
// Function to compute kernel
// ---------------------------------------------------------------------

/*! \fun void cuStenCompute2DXp
    \brief Function called by user to compute the stencil.
    \param pt_cuSten Pointer to cuSten data type which contains all the necessary input
	\param offload Set to HOST to move data back to CPU or DEVICE to keep on the GPU
*/

template <typename elemType>
void cuStenCompute2DXp
(
	cuSten_t<elemType>* pt_cuSten,
	bool offload
)
{	
	// Buffer used for error checking
	char msgStringBuffer[1024];

	// Set current active compute device
	hipSetDevice(pt_cuSten->deviceNum);
	sprintf(msgStringBuffer, "Setting current device to GPU %d", pt_cuSten->deviceNum);
	checkError(msgStringBuffer);

	dim3 blockDim(pt_cuSten->BLOCK_X, pt_cuSten->BLOCK_Y);
	dim3 gridDim(pt_cuSten->xGrid, pt_cuSten->yGrid);

	// Local memory grid sizes
	int local_nx = pt_cuSten->BLOCK_X + pt_cuSten->numStenLeft + pt_cuSten->numStenRight;
	int local_ny = pt_cuSten->BLOCK_Y;

	// Load the weights
	hipMemPrefetchAsync(pt_cuSten->weights, pt_cuSten->numSten * sizeof(double), pt_cuSten->deviceNum, pt_cuSten->streams[1]);

	// Preload the first block
	hipStreamSynchronize(pt_cuSten->streams[1]);

	// Prefetch the tile data
	hipMemPrefetchAsync(pt_cuSten->dataInput[0], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
	hipMemPrefetchAsync(pt_cuSten->dataOutput[0], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);

	// Record the event
	hipEventRecord(pt_cuSten->events[0], pt_cuSten->streams[1]);

	// Temporary stream and event used for permuting
	hipStream_t ts;
	hipEvent_t te;

	// Loop over the tiles
	for (int tile = 0; tile < pt_cuSten->numTiles; tile++)
	{
		// Synchronise the events to ensure computation overlaps
		hipEventSynchronize(pt_cuSten->events[0]);

		// Preform the computation on the current tile
		kernel2DXp<<<gridDim, blockDim, pt_cuSten->mem_shared, pt_cuSten->streams[0]>>>(pt_cuSten->dataOutput[tile], pt_cuSten->dataInput[tile], pt_cuSten->weights, pt_cuSten->numSten, pt_cuSten->numStenLeft, pt_cuSten->numStenRight, local_nx, local_ny, pt_cuSten->BLOCK_X, pt_cuSten->nx);

		// Offload should the user want to
		if (offload == 1)
		{
			hipMemPrefetchAsync(pt_cuSten->dataOutput[tile], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), hipCpuDeviceId, pt_cuSten->streams[0]);
	    	hipMemPrefetchAsync(pt_cuSten->dataInput[tile], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), hipCpuDeviceId, pt_cuSten->streams[0]);
		}

		// Load the next tile
    	if (tile < pt_cuSten->numTiles - 1)
    	{
    		// Ensure the steam is free to load the data
    		hipStreamSynchronize(pt_cuSten->streams[1]);

    		// Prefetch the necessary tiles  	
			hipMemPrefetchAsync(pt_cuSten->dataOutput[tile + 1], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
		 	hipMemPrefetchAsync(pt_cuSten->dataInput[tile + 1], pt_cuSten->nx * pt_cuSten->nyTile * sizeof(elemType), pt_cuSten->deviceNum, pt_cuSten->streams[1]);
	
			// Record the event
			hipEventRecord(pt_cuSten->events[1], pt_cuSten->streams[1]);
    	}

    	// Permute streams
    	for (int i = 0; i < pt_cuSten->numStreams - 1; i++)
    	{
    		ts = pt_cuSten->streams[i];
    		pt_cuSten->streams[i] = pt_cuSten->streams[i + 1];	
    		pt_cuSten->streams[i + 1] = ts;    			
    	}

    	// Permute events
		te = pt_cuSten->events[0]; pt_cuSten->events[0] = pt_cuSten->events[1]; pt_cuSten->events[1] = te; 
    }
}

// ---------------------------------------------------------------------
// Explicit instantiation
// ---------------------------------------------------------------------

template
__global__ void kernel2DXp<double>
(
	double*,  					
	double*,					
	const double*,       		
	const int,					
	const int,				
	const int,				
	const int,					
	const int,					
	const int,					
	const int
);

template
void cuStenCompute2DXp<double>
(
	cuSten_t<double>*,
	bool
);

template
__global__ void kernel2DXp<float>
(
	float*,  					
	float*,					
	const float*,       		
	const int,					
	const int,				
	const int,				
	const int,					
	const int,					
	const int,					
	const int
);

template
void cuStenCompute2DXp<float>
(
	cuSten_t<float>*,
	bool
);

// ---------------------------------------------------------------------
// End of file
// ---------------------------------------------------------------------
